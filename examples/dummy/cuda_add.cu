
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int a, int b, int * c)
{
  *c = a + b;
}


int main (void)
{
  int c;
  int * d_c;
  hipMalloc((void**)&d_c, sizeof(int));
  add<<<1, 1>>>(2, 7, d_c);
  hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);
  printf ("2+7=%d\n", c);
  hipFree (d_c);

  return 0;
}
